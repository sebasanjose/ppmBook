#include <stdio.h>
#include <hip/hip_runtime.h>

#define TILE_SIZE 16  // Define tile size for shared memory optimization

// Naïve matrix multiplication kernel
__global__ void matMulNaive(float *A, float *B, float *C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        float sum = 0.0;
        for (int k = 0; k < N; k++) {
            sum += A[row * N + k] * B[k * N + col];  // Multiple global memory accesses
        }
        C[row * N + col] = sum;
    }
}

// Optimized matrix multiplication using shared memory
__global__ void matMulTiled(float *A, float *B, float *C, int N) {
    __shared__ float A_shared[TILE_SIZE][TILE_SIZE];
    __shared__ float B_shared[TILE_SIZE][TILE_SIZE];

    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;
    float sum = 0.0;

    for (int t = 0; t < (N + TILE_SIZE - 1) / TILE_SIZE; t++) {
        // Load tiles into shared memory
        if (row < N && t * TILE_SIZE + threadIdx.x < N)
            A_shared[threadIdx.y][threadIdx.x] = A[row * N + t * TILE_SIZE + threadIdx.x];
        else
            A_shared[threadIdx.y][threadIdx.x] = 0.0;

        if (col < N && t * TILE_SIZE + threadIdx.y < N)
            B_shared[threadIdx.y][threadIdx.x] = B[(t * TILE_SIZE + threadIdx.y) * N + col];
        else
            B_shared[threadIdx.y][threadIdx.x] = 0.0;

        __syncthreads();  // Synchronize to ensure all threads have loaded data

        // Compute partial result
        for (int k = 0; k < TILE_SIZE; k++) {
            sum += A_shared[threadIdx.y][k] * B_shared[k][threadIdx.x];
        }

        __syncthreads();  // Ensure shared memory is not overwritten before next loop
    }

    if (row < N && col < N)
        C[row * N + col] = sum;
}

// Helper function to initialize matrices
void initializeMatrix(float *matrix, int N) {
    for (int i = 0; i < N * N; i++) {
        matrix[i] = static_cast<float>(rand() % 10);
    }
}

// Function to compare results
bool verifyResult(float *C1, float *C2, int N) {
    for (int i = 0; i < N * N; i++) {
        if (abs(C1[i] - C2[i]) > 1e-4) {
            printf("Mismatch at index %d: %f vs %f\n", i, C1[i], C2[i]);
            return false;
        }
    }
    return true;
}

int main() {
    int N = 512;  // Matrix size N x N
    size_t size = N * N * sizeof(float);

    // Allocate host memory
    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    float *h_C_naive = (float *)malloc(size);
    float *h_C_tiled = (float *)malloc(size);

    // Initialize matrices
    initializeMatrix(h_A, N);
    initializeMatrix(h_B, N);

    // Allocate device memory
    float *d_A, *d_B, *d_C_naive, *d_C_tiled;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C_naive, size);
    hipMalloc(&d_C_tiled, size);

    // Copy matrices to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 blockSize(TILE_SIZE, TILE_SIZE);
    dim3 gridSize((N + TILE_SIZE - 1) / TILE_SIZE, (N + TILE_SIZE - 1) / TILE_SIZE);

    // Run Naïve Kernel
    matMulNaive<<<gridSize, blockSize>>>(d_A, d_B, d_C_naive, N);
    hipMemcpy(h_C_naive, d_C_naive, size, hipMemcpyDeviceToHost);

    // Run Optimized Kernel
    matMulTiled<<<gridSize, blockSize>>>(d_A, d_B, d_C_tiled, N);
    hipMemcpy(h_C_tiled, d_C_tiled, size, hipMemcpyDeviceToHost);

    // Verify results
    if (verifyResult(h_C_naive, h_C_tiled, N)) {
        printf("Results match! Optimized kernel is correct.\n");
    } else {
        printf("Mismatch found! There is an error in the computation.\n");
    }

    // Free memory
    free(h_A);
    free(h_B);
    free(h_C_naive);
    free(h_C_tiled);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C_naive);
    hipFree(d_C_tiled);

    return 0;
}
