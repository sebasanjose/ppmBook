#include <iostream>
#include <hip/hip_runtime.h>

// CUDA Kernel function to add two arrays in parallel
__global__ void addArrays(int *a, int *b, int *c, int size) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < size) {
        c[index] = a[index] + b[index];
    }
}

int main() {
    int size = 1024;  // Size of arrays
    int bytes = size * sizeof(int);

    // Allocate host memory
    int *h_a = new int[size];
    int *h_b = new int[size];
    int *h_c = new int[size];

    // Initialize arrays
    for (int i = 0; i < size; i++) {
        h_a[i] = i;
        h_b[i] = i * 2;
    }

    // Allocate device memory
    int *d_a, *d_b, *d_c;
    hipMalloc((void**)&d_a, bytes);
    hipMalloc((void**)&d_b, bytes);
    hipMalloc((void**)&d_c, bytes);

    // Copy data from host to device
    hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

    // Define grid and block dimensions
    int threadsPerBlock = 256;   // Number of threads per block
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

    // Launch the kernel
    addArrays<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, size);

    // Copy result back to host
    hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

    // Print some results
    for (int i = 0; i < 10; i++) {
        std::cout << h_a[i] << " + " << h_b[i] << " = " << h_c[i] << std::endl;
    }

    // Free memory
    delete[] h_a; delete[] h_b; delete[] h_c;
    hipFree(d_a); hipFree(d_b); hipFree(d_c);

    return 0;
}
