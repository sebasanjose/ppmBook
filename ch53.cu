#include <stdio.h>
#include <hip/hip_runtime.h>

#define TILE_SIZE 16  // Tile size for shared memory optimization

// Naïve matrix multiplication (No tiling)
__global__ void matMulNaive(float *A, float *B, float *C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        float sum = 0.0;
        for (int k = 0; k < N; k++) {
            sum += A[row * N + k] * B[k * N + col];  // Multiple global memory accesses
        }
        C[row * N + col] = sum;
    }
}

// Optimized matrix multiplication using tiling
__global__ void matMulTiled(float *A, float *B, float *C, int N) {
    __shared__ float A_shared[TILE_SIZE][TILE_SIZE];
    __shared__ float B_shared[TILE_SIZE][TILE_SIZE];

    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;
    float sum = 0.0;

    for (int t = 0; t < (N + TILE_SIZE - 1) / TILE_SIZE; t++) {
        // Load tiles into shared memory
        if (row < N && t * TILE_SIZE + threadIdx.x < N)
            A_shared[threadIdx.y][threadIdx.x] = A[row * N + t * TILE_SIZE + threadIdx.x];
        else
            A_shared[threadIdx.y][threadIdx.x] = 0.0;

        if (col < N && t * TILE_SIZE + threadIdx.y < N)
            B_shared[threadIdx.y][threadIdx.x] = B[(t * TILE_SIZE + threadIdx.y) * N + col];
        else
            B_shared[threadIdx.y][threadIdx.x] = 0.0;

        __syncthreads();  // Ensure all threads have loaded data

        // Compute partial result
        for (int k = 0; k < TILE_SIZE; k++) {
            sum += A_shared[threadIdx.y][k] * B_shared[k][threadIdx.x];
        }

        __syncthreads();  // Ensure shared memory is not overwritten before next loop
    }

    if (row < N && col < N)
        C[row * N + col] = sum;
}

// Helper function to initialize matrices
void initializeMatrix(float *matrix, int N) {
    for (int i = 0; i < N * N; i++) {
        matrix[i] = static_cast<float>(rand() % 10);
    }
}

// Function to compare results
bool verifyResult(float *C1, float *C2, int N) {
    for (int i = 0; i < N * N; i++) {
        if (abs(C1[i] - C2[i]) > 1e-4) {
            printf("Mismatch at index %d: %f vs %f\n", i, C1[i], C2[i]);
            return false;
        }
    }
    return true;
}

int main() {
    int N = 512;  // Matrix size N x N
    size_t size = N * N * sizeof(float);

    // Allocate host memory
    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    float *h_C_naive = (float *)malloc(size);
    float *h_C_tiled = (float *)malloc(size);

    // Initialize matrices
    initializeMatrix(h_A, N);
    initializeMatrix(h_B, N);

    // Allocate device memory
    float *d_A, *d_B, *d_C_naive, *d_C_tiled;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C_naive, size);
    hipMalloc(&d_C_tiled, size);

    // Copy matrices to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 blockSize(TILE_SIZE, TILE_SIZE);
    dim3 gridSize((N + TILE_SIZE - 1) / TILE_SIZE, (N + TILE_SIZE - 1) / TILE_SIZE);

    // Measure time for Naïve Kernel
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    matMulNaive<<<gridSize, blockSize>>>(d_A, d_B, d_C_naive, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float naiveTime = 0;
    hipEventElapsedTime(&naiveTime, start, stop);

    hipMemcpy(h_C_naive, d_C_naive, size, hipMemcpyDeviceToHost);

    // Measure time for Optimized Tiled Kernel
    hipEventRecord(start);
    matMulTiled<<<gridSize, blockSize>>>(d_A, d_B, d_C_tiled, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float tiledTime = 0;
    hipEventElapsedTime(&tiledTime, start, stop);

    hipMemcpy(h_C_tiled, d_C_tiled, size, hipMemcpyDeviceToHost);

    // Verify results
    if (verifyResult(h_C_naive, h_C_tiled, N)) {
        printf("Results match! Optimized kernel is correct.\n");
    } else {
        printf("Mismatch found! There is an error in the computation.\n");
    }

    // Print performance comparison
    printf("\nMatrix Multiplication Performance Comparison\n");
    printf("--------------------------------------------\n");
    printf("| Method       | Execution Time (ms) |\n");
    printf("|-------------|--------------------|\n");
    printf("| Naïve       | %10.4f ms        |\n", naiveTime);
    printf("| Tiled       | %10.4f ms        |\n", tiledTime);
    printf("--------------------------------------------\n");

    // Free memory
    free(h_A);
    free(h_B);
    free(h_C_naive);
    free(h_C_tiled);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C_naive);
    hipFree(d_C_tiled);

    return 0;
}
